
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define EPS (0.0000001f)

#define M (4*1048576)
__device__ int gl[M];


__device__ float Cp[M]; 


// Thomas resolution for tridiagonal symmetric matrices
__global__ void thom_sym_k(float *S, float *D, float *Y, int n){

	// The global memory access index
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int j;

	float d = D[idx*n];
	Cp[idx*n + 1] = S[idx*n + 1] / d; 
	Y[idx*n] = S[idx*n + 1] / d;

	for (j=1;j<n;j++) { //Decomposition and forward substitution.
		Cp[idx*n + j + 1] = S[idx*n + j + 1] / (D[idx*n + j] - S[idx*n + j + 1]*Cp[idx*n + j]);
		Y[idx*n + j] = (Y[idx*n + j] - S[idx*n + j + 1]*Y[idx*n + j - 1]) / (D[idx*n + j] - S[idx*n + j + 1]*Cp[idx*n + j]);
	}

	for (j=(n-2);j>=0;j--){ //Backsubstitution.
		Y[idx*n + j] = Y[idx*n + j] - Cp[idx*n + j + 1]*Y[idx*n + j + 1];
	}
}

// Parallel cyclic reduction for tridiagonal symmetric matrices
__global__ void pcr_sym_k(float *a, float *b, float *y, int n)
{
	// Identifies the thread working within a group
	int tidx = threadIdx.x%n;

	// Identifies the data concerned by the computations
	int Qt = (threadIdx.x - tidx) / n;

	// The global memory access index
	int gb_index_x = Qt + blockIdx.x*(blockDim.x / n);

	int i; 
	int nt = 4*Qt*n;
	int d = (n / 2 + (n % 2))*(tidx % 2) + (int)tidx / 2;
	
	// Shared memory
	extern __shared__ float sAds[];

	float *sa = (float*)&sAds[nt];
	float *sb = (float*)&sa[n];
	float *sy = (float*)&sb[n];
	int *sl = (int*)&sy[n];

	sa[tidx] = a[gb_index_x*n + tidx];
	sb[tidx] = b[gb_index_x*n + tidx];
	sy[tidx] = y[gb_index_x*n + tidx];
	sl[tidx] = tidx;

	int lL, aL, bL, yL, bLp, tl, tr; // local variables

	//////////////////////////////////////////////////////////////
	//
	//	Step 2:	Fill with your code : Additional variables definition 
	//						  and copy the values in shared 
	//
	//////////////////////////////////////////////////////////////
	__syncthreads();

	tl = tidx-1;
	tr = tidx+1;
	if(tl<0) tl=0;
	if(tr>=n) tr=0;

	for (i = 0; i < (int)log2((float)n) + 1; i++){
		lL = sl[tidx];
		aL = sa[tidx];
		bL = sb[tidx];
		yL = sy[tidx];
		bLp = sb[tl];


		//Reduction phase
		//////////////////////////////////////////////////////////////
		//
		//		Step 2:	Fill with your code
		//
		//////////////////////////////////////////////////////////////
		__syncthreads();
		if (i < (int)log2((float)n)){//Permutation phase
			sa[d] = aL;
			sa[d] = bL;
			sa[d] = yL;
			sa[d] = lL;
			__syncthreads();
		}
	}

	//////////////////////////////////////////////////////////////
	// écrire la solution de shared vers la mémoire globale 
	//////////////////////////////////////////////////////////////
}


// Produces tridiagonal symmetric diagonally dominant matrices 
__global__ void Tri_k(float *D, float *S, float norm, int i, 
						   int n, int L)
{
	// Identifies the thread working within a group
	int tidx = threadIdx.x%n;
	// Identifies the data concerned by the computations
	int Qt = (threadIdx.x - tidx) / n;
	// The global memory access index
	int gb_index_x = Qt + blockIdx.x*(blockDim.x / n);

	if(gb_index_x*n + tidx + i < L){
		D[gb_index_x*n + tidx + i] = ((float)tidx+1.0f)/(norm);
		if (tidx > 0){
			S[gb_index_x*n + tidx + i] = ((float)tidx+1.0f)/(norm*3);
		}else{S[gb_index_x*n + tidx + i] = 0.0f;}
	}
}


int main(){

	int i, j;

	// The rank of the matrix
	int Dim = 64;
	
	// The number of blocks
	int NB = M/Dim;
	
	// The number of matrices to invert
	int size = NB;

	// The diagonal elements
	float *D, *DGPU;
	// The subdiagonal elements
	float *S, *SGPU;
	// The system vector
	float *Y, *YGPU;

	float TimerV;					// GPU timer instructions
	hipEvent_t start, stop;		// GPU timer instructions
	hipEventCreate(&start);		// GPU timer instructions
	hipEventCreate(&stop);			// GPU timer instructions

	// Memory allocation
	D = (float *)calloc(size*Dim,sizeof(float));
	S = (float *)calloc(size*Dim,sizeof(float));
	Y = (float *)calloc(size*Dim,sizeof(float));
	hipMalloc(&DGPU, size*Dim*sizeof(float));
	hipMalloc(&SGPU, size*Dim*sizeof(float));
	hipMalloc(&YGPU, size*Dim*sizeof(float));

	// Tridiagonal elements
	int HM = M/(NB*Dim);
	for (i=0; i*Dim*NB<M; i++){
		Tri_k <<<NB,HM*Dim>>>(DGPU, SGPU, 10.0f, i*Dim*NB, Dim, 
										  Dim*NB);
	}

	hipMemcpy(D, DGPU, size*Dim*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(S, SGPU, size*Dim*sizeof(float), hipMemcpyDeviceToHost);

	// Second member
	for (i=0; i<size; i++){
		for (j=0; j<Dim; j++){
			Y[j+i*Dim]=0.5f*j;
		}
	}
	hipMemcpy(YGPU,Y,size*Dim*sizeof(float),hipMemcpyHostToDevice);


	// Resolution part
	hipEventRecord(start,0);


	/////////////////////////////////////////////////////////////////////
	// Step 2:	PCR
	/////////////////////////////////////////////////////////////////////
	// The minimum group of threads per block for PCR /!\ Has to be chosen by students
	// /!\ /!\ /!\ /!\ /!\ /!\ /!\ /!\ /!\ /!\

	/*
	int minTB = 1;  // Choose the right value with respect to Dim
	printf("%i \n", minTB);
	pcr_sym_k<<<NB/minTB,Dim*minTB,4*minTB*Dim*sizeof(float)>>>(SGPU, DGPU, YGPU, Dim);
	*/

	/////////////////////////////////////////////////////////////////////
	// Step 1:	Thomas
	/////////////////////////////////////////////////////////////////////
	thom_sym_k<<<NB/256,256>>>(SGPU, DGPU, YGPU, Dim);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&TimerV,start, stop);

	hipMemcpy(Y, YGPU, size*Dim*sizeof(float), hipMemcpyDeviceToHost);
	for (i=0; i<size; i++){
	        if(i==573){
			printf("\n\n");
			for (j=0; j<Dim; j++){
				printf("%.5e, ",Y[j+i*Dim]);
			}
		}
	}


	printf("Execution time: %f ms\n", TimerV);

	// Memory free for other arrays
	free(D);
	hipFree(DGPU);
	free(S);
	hipFree(SGPU);
	free(Y);
	hipFree(YGPU);

	hipEventDestroy(start);		// GPU timer instructions
	hipEventDestroy(stop);			// GPU timer instructions

	return 0;
}
