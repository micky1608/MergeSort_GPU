
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#define EPS (0.0000001f)

#define M (4*1048576)
__device__ int gl[M];


__device__ float Cp[M]; 


// Thomas resolution for tridiagonal symmetric matrices
__global__ void thom_sym_k(float *S, float *D, float *Y, int n){

	// The global memory access index
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int j;


	int first = idx*n;
	int first_c = idx*n + 1;

	float d = D[first];
	Cp[first_c] = S[first_c] / d; 
	Y[first] = Y[first] / d;

	for (j=1;j<n-1;j++) { //Decomposition and forward substitution.
		Cp[first_c + j] = S[first_c + j] / (D[first + j] - S[first_c + j - 1]*Cp[first_c + j - 1]);	
		Y[first + j] = (Y[first + j] - S[first_c + j - 1]*Y[first + j - 1]) / (D[first + j] - S[first_c + j - 1]*Cp[first_c + j - 1]);
	}

	// One more iteration for Y (j=n-1)
	Y[first + n - 1] = (Y[first + n - 1] - S[first_c + n - 2]*Y[first + n - 2]) / (D[first + n - 1] - S[first_c + n - 2]*Cp[first_c + n - 2]);

	for (j=(n-2);j>=0;j--){ //Backsubstitution.
		Y[first + j] = Y[first + j] - Cp[first_c + j]*Y[first + j + 1];
	}
}

///////////////////////////////////////////////////////////////////////
// Parallel cyclic reduction for tridiagonal symmetric matrices
///////////////////////////////////////////////////////////////////////
__global__ void pcr_sym_k(float *a, float *b, float *y, int n)
{
	// Identifies the thread working within a group
	int tidx = threadIdx.x%n;
	// Identifies the data concerned by the computations
	int Qt = (threadIdx.x - tidx) / n;
	// The global memory access index
	int gb_index_x = Qt + blockIdx.x*(blockDim.x / n);
	printf("truc\n");

	int i;
	int nt = 4 * Qt*n;
	int d = (n / 2 + (n % 2))*(tidx % 2) + (int)tidx / 2;
	// Shared memory
	extern __shared__ float sAds[];


	float *sa = (float*)&sAds[nt];
	float *sb = (float*)&sa[n];
	float *sy = (float*)&sb[n];
	int *sl = (int*)&sy[n];

	sa[tidx] = a[gb_index_x*n + tidx];
	sb[tidx] = b[gb_index_x*n + tidx];
	sy[tidx] = y[gb_index_x*n + tidx];
	sl[tidx] = tidx;

	if (threadIdx.x == 0) {
		printf("Values : \n");
		for (i = 0; i < 4 * 2 *n; ++i) {
			printf("%.5e, ", sAds[i]);
		}
	}

	int lL, aL, bL, yL, bLp, tl, tr; // local variables
	float aLp, yLp;

	__syncthreads();

	tl = tidx - 1;
	tr = tidx + 1;
	if (tl < 0) tl = 0;
	if (tr >= n) tr = 0;

	for (i = 0; i < (int)log2((float)n) + 1; i++) {
		lL = sl[tidx];
		aL = sa[tidx];
		bL = sb[tidx];
		yL = sy[tidx];
		bLp = sb[tl];


		if (fabsf(aL) > EPS) {
			aLp = sa[tl];
			yLp = sy[tl];

			bL -= aL * aL / bLp;
			yL -= aL * yLp / bLp;
			aL = -aL * aLp / bLp;
		}

		aLp = sa[tr];
		bLp = sb[tr];
		if (fabsf(aLp) > EPS) {
			yLp = sy[tr];
			bL -= aLp * aLp / bLp;
			yL -= aLp * yLp / bLp;
		}

		__syncthreads();
		if (i < (int)log2((float)n)) {//Permutation phase
			sa[d] = aL;
			sb[d] = bL;
			sy[d] = yL;
			sl[d] = lL;
			__syncthreads();
		}
	}

	sy[(int)tidx] = yL / bL;
	__syncthreads();

	float sum = 0.0f;
	//Second matrix/vector product 
	//Non-coalescent access to recover the solution
	for (i = 0; i < n; i++) {
		sum += sa[tidx*n + (int)sl[i]] * sy[i];
	}
	y[gb_index_x*n + tidx] = sum;
}

// Parallel cyclic reduction for tridiagonal symmetric matrices
__global__ void pcr_sym_k3(float *a, float *b, float *y, int n)
{
	// Identifies the thread working within a group
	int tidx = threadIdx.x%n;

	// Identifies the data concerned by the computations
	int Qt = (threadIdx.x - tidx) / n;

	// The global memory access index
	int gb_index_x = Qt + blockIdx.x*(blockDim.x / n);

	int i; 
	int nt = 4*Qt*n;
	int d = (n / 2 + (n % 2))*(tidx % 2) + (int)tidx / 2;
	
	// Shared memory
	extern __shared__ float sAds[];

	float *sa = (float*)&sAds[nt];
	float *sb = (float*)&sa[n];
	float *sy = (float*)&sb[n];
	int *sl = (int*)&sy[n];

	sa[tidx] = a[gb_index_x*n + tidx];
	sb[tidx] = b[gb_index_x*n + tidx];
	sy[tidx] = y[gb_index_x*n + tidx];
	sl[tidx] = tidx;

	int lL, aL, bL, yL, bLp, tl, tr; // local variables

	//////////////////////////////////////////////////////////////
	//
	//	Step 2:	Fill with your code : Additional variables definition 
	//						  and copy the values in shared 
	//
	//////////////////////////////////////////////////////////////
	// Local floats
	float aLp, yLp;

	/*sum = 0.0f;
	for (i = 0; i < n; i++) {
		sq[i*n + tidx] = q[gb_index_x*n2 + i * n + tidx];
		sum += sq[i*n + tidx] * sy[i];
	}
	sy[tidx] = sum;*/
	__syncthreads();

	tl = tidx-1;
	tr = tidx+1;
	if(tl<0) tl=0;
	if(tr>=n) tr=0;

	for (i = 0; i < (int)log2((float)n) + 1; i++){
		lL = sl[tidx];
		aL = sa[tidx];
		bL = sb[tidx];
		yL = sy[tidx];
		bLp = sb[tl];


		if (fabsf(aL) > EPS) {
			aLp = sa[tl];
			yLp = sy[tl];

			bL -= aL * aL / bLp;
			yL -= aL * yLp / bLp;
			aL = -aL * aLp / bLp;
		}

		aLp = sa[tr];
		bLp = sb[tr];
		if (fabsf(aLp) > EPS) {
			yLp = sy[tr];
			yL -= aLp * yLp / bLp;
			bL -= aLp * aLp / bLp;
		}

		__syncthreads();
		//Permutation phase
		if (i < (int)log2((float)n)) {
			sa[d] = aL;
			sb[d] = bL;
			sy[d] = yL;
			sl[d] = (int)lL;
			__syncthreads();
		}
	}

	//////////////////////////////////////////////////////////////
	// écrire la solution de shared vers la mémoire globale 
	//////////////////////////////////////////////////////////////

	sy[(int)tidx] = yL / bL;
	__syncthreads();

	float sum = 0.0f;
	//Second matrix/vector product 
	//Non-coalescent access to recover the solution
	for (i = 0; i < n; i++) {
		sum += sa[tidx*n + (int)sl[i]] * sy[i];
	}
	y[gb_index_x*n + tidx] = sum;
}

__global__ void pcr_sym_k2(float *q, float *a, float *b, float *y, int n)
{
	// Identifies the thread working within a group
	int tidx = threadIdx.x%n;
	// Identifies the data concerned by the computations
	int Qt = (threadIdx.x - tidx) / n;
	// The global memory access index
	int gb_index_x = Qt + blockIdx.x*(blockDim.x / n);
	// Local integers
	int i, n2, nt, lL, d, tL, tR;
	// Local floats
	float aL, bL, yL, sum, aLp, bLp, yLp;
	// Shared memory
	extern __shared__ float sAds[];

	n2 = n * n;
	nt = Qt * n*(n + 4);
	d = (n / 2 + (n % 2))*(tidx % 2) + (int)tidx / 2;

	float* sq = (float*)&sAds[nt];
	float* sa = (float*)&sq[n2];
	float* sb = (float*)&sa[n];
	float* sy = (float*)&sb[n];
	int* sl = (int*)&sy[n];

	sa[tidx] = a[gb_index_x*n + tidx];
	sb[tidx] = b[gb_index_x*n + tidx];
	sy[tidx] = y[gb_index_x*n + tidx];
	sl[tidx] = tidx;
	__syncthreads();

	//First matrix/vector product
	sum = 0.0f;
	for (i = 0; i < n; i++) {
		sq[i*n + tidx] = q[gb_index_x*n2 + i * n + tidx];
		sum += sq[i*n + tidx] * sy[i];
	}
	__syncthreads();
	sy[tidx] = sum;

	//Left/Right indices of the reduction
	tL = tidx - 1;
	if (tL < 0) tL = 0;
	tR = tidx + 1;
	if (tR >= n) tR = 0;

	for (i = 0; i < (int)log2((float)n) + 1; i++) {
		lL = (int)sl[tidx];

		aL = sa[tidx];
		bL = sb[tidx];
		yL = sy[tidx];

		bLp = sb[tL];

		//Reduction phase
		if (fabsf(aL) > EPS) {
			aLp = sa[tL];
			yLp = sy[tL];

			//bL = b[tidx] - a[tidx]*c[tidx]/b[tidx-1];
			bL -= aL * aL / bLp;
			//yL = y[tidx] - a[tidx]*y[tidx-1]/b[tidx-1];
			yL -= aL * yLp / bLp;
			//aL = -a[tidx]*a[tidx-1]/b[tidx-1];
			aL = -aL * aLp / bLp;
		}

		aLp = sa[tR];
		bLp = sb[tR];
		if (fabsf(aLp) > EPS) {
			yLp = sy[tR];
			//bL -= c[tidx+1]*a[tidx+1]/b[tidx+1];
			bL -= aLp * aLp / bLp;
			//yL -= c[tidx+1]*y[tidx+1]/b[tidx+1];
			yL -= aLp * yLp / bLp;
		}
		__syncthreads();

		//Permutation phase
		if (i < (int)log2((float)n)) {
			sa[d] = aL;
			sb[d] = bL;
			sy[d] = yL;
			sl[d] = (int)lL;
			__syncthreads();
		}
	}

	sy[(int)tidx] = yL / bL;
	__syncthreads();

	sum = 0.0f;
	//Second matrix/vector product 
	//Non-coalescent access to recover the solution
	for (i = 0; i < n; i++) {
		sum += sq[tidx*n + (int)sl[i]] * sy[i];
	}
	y[gb_index_x*n + tidx] = sum;
}


// Produces tridiagonal symmetric diagonally dominant matrices 
__global__ void Tri_k(float *D, float *S, float norm, int i, 
						   int n, int L)
{
	// Identifies the thread working within a group
	int tidx = threadIdx.x%n;
	// Identifies the data concerned by the computations
	int Qt = (threadIdx.x - tidx) / n;
	// The global memory access index
	int gb_index_x = Qt + blockIdx.x*(blockDim.x / n);

	if(gb_index_x*n + tidx + i < L){
		D[gb_index_x*n + tidx + i] = ((float)tidx+1.0f)/(norm);
		if (tidx > 0){
			S[gb_index_x*n + tidx + i] = ((float)tidx+1.0f)/(norm*3);
		}else{S[gb_index_x*n + tidx + i] = 0.0f;}
	}
}


int main(){

	int i, j;

	// The rank of the matrix
	int Dim = 64;
	
	// The number of blocks
	int NB = M/Dim;
	
	// The number of matrices to invert
	int size = NB;

	// The diagonal elements
	float *D, *DGPU;
	// The subdiagonal elements
	float *S, *SGPU;
	// The system vector
	float *Y, *YGPU;
	float *A, *AGPU;

	float TimerV;					// GPU timer instructions
	hipEvent_t start, stop;		// GPU timer instructions
	hipEventCreate(&start);		// GPU timer instructions
	hipEventCreate(&stop);			// GPU timer instructions

	// Memory allocation
	A = (float *)calloc(size*Dim*Dim, sizeof(float));
	D = (float *)calloc(size*Dim,sizeof(float));
	S = (float *)calloc(size*Dim,sizeof(float));
	Y = (float *)calloc(size*Dim,sizeof(float));
	hipMalloc(&AGPU, size*Dim*Dim * sizeof(float));
	hipMalloc(&DGPU, size*Dim*sizeof(float));
	hipMalloc(&SGPU, size*Dim*sizeof(float));
	hipMalloc(&YGPU, size*Dim*sizeof(float));

	// Tridiagonal elements
	int HM = M/(NB*Dim);
	for (i=0; i*Dim*NB<M; i++){
		Tri_k <<<NB,HM*Dim>>>(DGPU, SGPU, 10.0f, i*Dim*NB, Dim, 
										  Dim*NB);
	}

	hipMemcpy(D, DGPU, size*Dim*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(S, SGPU, size*Dim*sizeof(float), hipMemcpyDeviceToHost);

	for (i = 0; i < Dim; ++i) {
		for (j = 0; j < Dim; ++j) {
			if (i == j)
				A[i*Dim + j] = D[i];
			else if (j - 1 == i)
				A[i*Dim + j] = S[j - 1];
			else if (j + 1 == i)
				A[i*Dim + j] = S[j];
			else
				A[i*Dim + j] = 0.0f;
		}
	}

	// Second member
	for (i=0; i<size; i++){
		for (j=0; j<Dim; j++){
			Y[j+i*Dim]=0.5f*j;
		}
	}
	hipMemcpy(AGPU, A, size*Dim*Dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(YGPU,Y,size*Dim*sizeof(float),hipMemcpyHostToDevice);


	// Resolution part
	hipEventRecord(start,0);


	/////////////////////////////////////////////////////////////////////
	// Step 2:	PCR
	/////////////////////////////////////////////////////////////////////
	// The minimum group of threads per block for PCR /!\ Has to be chosen by students
	// /!\ /!\ /!\ /!\ /!\ /!\ /!\ /!\ /!\ /!\
	
	int minTB = 2;  // Choose the right value with respect to Dim
	printf("%i \n", minTB);
	//pcr_sym_k<<<NB/minTB, Dim*minTB, 4*minTB*Dim*sizeof(float)>>>(SGPU, DGPU, YGPU, Dim);
	//pcr_sym_k2 << <NB, Dim*minTB, (minTB*Dim*(Dim + 4)) * sizeof(float) >> > (AGPU, SGPU, DGPU, YGPU, Dim);
	

	/////////////////////////////////////////////////////////////////////
	// Step 1:	Thomas
	/////////////////////////////////////////////////////////////////////
	thom_sym_k<<<NB/256,256>>>(SGPU, DGPU, YGPU, Dim);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&TimerV,start, stop);

	hipMemcpy(Y, YGPU, size*Dim*sizeof(float), hipMemcpyDeviceToHost);
	for (i=0; i<size; i++){
	        if(i==573){
			printf("\n\n");
			for (j=0; j<Dim; j++){
				printf("%.5e, ",Y[j+i*Dim]);
			}
		}
	}


	printf("Execution time: %f ms\n", TimerV);

	// Memory free for other arrays
	free(A);
	hipFree(AGPU);
	free(D);
	hipFree(DGPU);
	free(S);
	hipFree(SGPU);
	free(Y);
	hipFree(YGPU);

	hipEventDestroy(start);		// GPU timer instructions
	hipEventDestroy(stop);			// GPU timer instructions

	return 0;
}