#include "hip/hip_runtime.h"

#include "cuda_header.h"

__device__ int A_diag[D];
__device__ int B_diag[D];


__global__ void mergeSmall_k(int *A, int length_A, int *B, int length_B, int *M) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	duo K, P, Q;
	if (i > length_A) {
			K.x = i - length_A;
			K.y = length_A;
			P.x = length_A;
			P.y = i - length_A;
	}else {
			K.x = 0;
			K.y = i;
			P.x = i;
			P.y = 0;
	}
	while (true) {
		int offset = abs(K.y - P.y) / 2;
		Q.x = K.x + offset;
		Q.y = K.y - offset;
		if (Q.y >= 0 && Q.x <= length_B && (Q.y == length_A || Q.x == 0 || A[Q.y] > B[Q.x - 1])) {
				if (Q.x == length_B || Q.y == 0 || A[Q.y - 1] <= B[Q.x]) {
						if (Q.y < length_A && (Q.x == length_B || A[Q.y] <= B[Q.x])) {
								M[i] = A[Q.y];
						}else {
								M[i] = B[Q.x];
						}
						break;
				}
				else {
					K.x = Q.x + 1;
					K.y = Q.y - 1;
				}
		}
		else {
			P.x = Q.x - 1;
			P.y = Q.y + 1;
		}
	}
}


__global__ void mergeSmallBatch_k(int *AB, int length_A, int length_B, int *M, int nb_merge) {
	int nb_threads = gridDim.x * blockDim.x,
		gtidx = threadIdx.x + blockIdx.x * blockDim.x,
		d = length_A+length_B;
	
	int t, tidx, qt, gbx, begin_M, begin_A, begin_B;

	duo K, P, Q;

	while(gtidx < nb_merge*d) {
		
		t = gtidx % blockDim.x,                   		// index of the thread inside a block : 0 -> blockDim.x - 1
		tidx = t%d,                             		// index of the thread in its corresponding final array : 0 -> d-1
		qt = (t-tidx)/d,                        		// index of the group of the thread inside a block : 0 -> (blockDim.x/d)-1
		gbx = (blockDim.x/d)*blockIdx.x + qt,   		// index of the group of the thread among all the blocks : 0 -> (blockDim.x/d)*gridDim.x - 1
		begin_M = gbx*d,                        		// index of the first element of M
		begin_A = gbx*d,                 				// index of the first element of A
		begin_B = begin_A + length_A;                 	// index of the first element of B

		if (tidx > length_A) {
				K.x = tidx - length_A;
				K.y = length_A;
				P.x = length_A;
				P.y = tidx - length_A;
		}else {
				K.x = 0;
				K.y = tidx;
				P.x = tidx;
				P.y = 0;
		}
		while (true) {
				int offset = abs(K.y - P.y) / 2;
				Q.x = K.x + offset;
				Q.y = K.y - offset;
				if (Q.y >= 0 && Q.x <= length_B && (Q.y == length_A || Q.x == 0 || AB[begin_A + Q.y] > AB[begin_B + Q.x - 1])) {
						if (Q.x == length_B || Q.y == 0 || AB[begin_A + Q.y - 1] <= AB[begin_B + Q.x]) {
								if (Q.y < length_A && (Q.x == length_B || AB[begin_A + Q.y] <= AB[begin_B + Q.x])) {
										M[begin_M + tidx] = AB[begin_A + Q.y];
								}else {
										M[begin_M + tidx] = AB[begin_B + Q.x];
								}
								break;
						}
						else {
						K.x = Q.x + 1;
						K.y = Q.y - 1;
						}
				}
				else {
				P.x = Q.x - 1;
				P.y = Q.y + 1;
				}
		}
		gtidx += nb_threads;
	}
}

/**
	Find the intersection between one diagonal and the merge path.
*/
__global__ void pathBig_k(int *A, int length_A, int *B, int length_B, int start_diag) {
	int nb_threads = gridDim.x * blockDim.x;
	int tidx = threadIdx.x + blockIdx.x*blockDim.x;
	int length = (length_A+length_B)/nb_threads;
	

	duo K, P, Q;

	while(tidx<(length_A+length_B)) {
		int index_diag = start_diag + tidx*length;

		for(int k=0 ; k<length; ++k) {
			int i = tidx*length+k;
				if (i > length_A) {
						K.x = i - length_A;
						K.y = length_A;
						P.x = length_A;
						P.y = i - length_A;
				}else {
						K.x = 0;
				K.y = i;
				P.x = i;
				P.y = 0;
			}
		
			while (true) {
				int offset = abs(K.y - P.y) / 2;
				Q.x = K.x + offset;
				Q.y = K.y - offset;
				if (Q.y >= 0 && Q.x <= length_B && (Q.y == length_A || Q.x == 0 || A[Q.y] > B[Q.x - 1])) {
					if (Q.x == length_B || Q.y == 0 || A[Q.y - 1] <= B[Q.x]) {
						A_diag[index_diag+k] = Q.y;
						B_diag[index_diag+k] = Q.x;
						break;
					}
					else {
						K.x = Q.x + 1;
						K.y = Q.y - 1;
					}
				}
				else {
					P.x = Q.x - 1;
					P.y = Q.y + 1;
				}
			}
		}
		tidx += nb_threads;
	}
	
}

__global__ void mergeBig_k(int *A, int length_A, int *B, int length_B, int* M, int start_diag) {
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;

	int nb_threads = gridDim.x * blockDim.x;
	int length = (length_A + length_B) / nb_threads;
	
	while(tidx<(length_A+length_B)) {
		int start_M = start_diag + tidx*length;

		for(int k=0 ; k<length ; ++k) {
				int i = start_M + k;
			if (A_diag[i] < length_A && (B_diag[i] == length_B || A[A_diag[i]] <= B[B_diag[i]])) {
				M[i] = A[A_diag[i]];
			} else {
				M[i] = B[B_diag[i]];
			}
		}
		tidx += nb_threads;
	}
}


void mergeSortGPU (int *M , int length, float *timer) {
    float timer_iter;
    int *M_dev, *M_dev_copy;
    int merge_size = 2;
    hipEvent_t start, stop, start_iter, stop_iter;

    testCUDA(hipEventCreate(&start));
    testCUDA(hipEventCreate(&stop));
    testCUDA(hipEventCreate(&start_iter));
    testCUDA(hipEventCreate(&stop_iter));

    testCUDA(hipMalloc((void**)&M_dev , D*sizeof(int)));
    testCUDA(hipMalloc((void**)&M_dev_copy , D*sizeof(int)));
    testCUDA(hipMemcpy(M_dev, M,D*sizeof(int), hipMemcpyHostToDevice));

    testCUDA(hipEventRecord(start, 0));
    
    while(merge_size <= pow(2,ceil(log2(length)))) {
	
    	testCUDA(hipEventRecord(start_iter,0));

		testCUDA(hipMemcpy(M_dev_copy, M_dev, D * sizeof(int), hipMemcpyDeviceToDevice));
	
		if(merge_size <= BATCH_THRESHOLD) {
			int block_size =  min(length, ((int)(1024/merge_size))*merge_size);
			int nb_block = (length + block_size - 1)/block_size;
			mergeSmallBatch_k<<<nb_block,block_size>>>(M_dev_copy, merge_size/2, merge_size/2, M_dev, length/merge_size);

			if(length%merge_size) {
				int merge_size_last = length%merge_size;
				int block_size_last = (merge_size_last > 1024) ? 1024 : merge_size_last;
				int nb_block_last = (merge_size_last + block_size_last - 1)/block_size_last;
				
				if(merge_size_last > merge_size/2) 
					mergeSmall_k<<<nb_block_last,block_size_last>>>(M_dev_copy + ((int)(length/merge_size))*merge_size, merge_size/2, M_dev_copy + ((int)(length/merge_size))*merge_size + (merge_size/2), merge_size_last-(merge_size/2), M_dev + ((int)(length/merge_size))*merge_size);
			}
		}
		else {
			int iter = (length+merge_size-1)/merge_size;
			int block_size = (merge_size > 1024) ? 1024 : merge_size;
			int nb_block = (merge_size+block_size-1)/block_size;
			
			for (int k = 0; k < iter; ++k) {
				if(k<(length/merge_size))
					pathBig_k << <nb_block,block_size>> > (M_dev + k * merge_size, merge_size / 2, M_dev + (2 * k + 1)*(merge_size / 2), merge_size / 2, merge_size*k);
				else {
					int merge_size_last = length%merge_size;
					int block_size_last = (merge_size_last > 1024) ? 1024 : merge_size_last;
					int nb_block_last = (merge_size_last + block_size_last - 1)/block_size_last;
					
					while(nb_block_last*block_size_last > merge_size_last) block_size_last /= 2;
					
					if(merge_size_last > merge_size/2) 
						pathBig_k << <nb_block_last, block_size_last >> > (M_dev + k * merge_size, merge_size / 2,M_dev + k * merge_size + merge_size/2, merge_size_last - (merge_size/2), merge_size*k);
				}
			}
			testCUDA(hipDeviceSynchronize());
				
			for(int k=0 ; k<iter ; ++k) {
				if(k<(length/merge_size))
					mergeBig_k<<<nb_block,block_size>>>(M_dev_copy+k*merge_size, merge_size/2, M_dev_copy+(2*k+1)*(merge_size/2), merge_size/2, M_dev, merge_size*k);
				else {
					int merge_size_last = length%merge_size;
					int block_size_last = (merge_size_last > 1024) ? 1024 : merge_size_last;
					int nb_block_last = (merge_size_last + block_size_last - 1)/block_size_last;
					
					while(nb_block_last*block_size_last > merge_size_last) block_size_last /= 2;
		
					if(merge_size_last > merge_size/2)
						mergeBig_k << <nb_block_last, block_size_last >> > (M_dev_copy + k * merge_size, merge_size / 2,M_dev_copy + k * merge_size + merge_size/2, merge_size_last - (merge_size/2),  M_dev, merge_size*k);
				
				}
		
			}
		}

		testCUDA(hipDeviceSynchronize());
		testCUDA(hipEventRecord(stop_iter,0));
		testCUDA(hipEventSynchronize(stop_iter));
		testCUDA(hipEventElapsedTime(&timer_iter,start_iter, stop_iter));
		printf("MergeSize = %7d\t\tDuration : %f ms\n",merge_size,timer_iter);
		merge_size *= 2;

	}

    testCUDA(hipEventRecord(stop, 0));
    testCUDA(hipEventSynchronize(stop));
    testCUDA(hipEventElapsedTime(timer, start, stop));
    
    testCUDA(hipMemcpy(M, M_dev,D*sizeof(int), hipMemcpyDeviceToHost));
    testCUDA(hipFree(M_dev));
    testCUDA(hipFree(M_dev_copy));

    testCUDA(hipEventDestroy(start));
    testCUDA(hipEventDestroy(stop));
    testCUDA(hipEventDestroy(start_iter));
    testCUDA(hipEventDestroy(stop_iter));

}

int main(int argc , char *argv[]) {
    // initialize random seed
    srand(time(0));
    float TimerAdd = 0;

    printf("Size of array : %d\n",D);

    int* M = generate_unsorted_array(D);
    //print_unsorted_array(M , D , "M");

    mergeSortGPU(M,D, &TimerAdd);

    check_array_sorted(M,D,"M");
    printf("===== Total time : %f ms =====\n", TimerAdd);
    free(M);
}
